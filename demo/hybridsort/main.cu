#include "hip/hip_runtime.h"
#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"
#include <iostream>
#include <sys/time.h>
#include "bucketsort.cuh"
#include "mergesort.cuh"

using namespace std; 

////////////////////////////////////////////////////////////////////////////////
// Size of the testset (Bitwise shift of 1 over 22 places)
////////////////////////////////////////////////////////////////////////////////
#define SIZE	(1 << 22)
////////////////////////////////////////////////////////////////////////////////
// Number of tests to average over
////////////////////////////////////////////////////////////////////////////////
#define TEST	1
////////////////////////////////////////////////////////////////////////////////
// The timers for the different parts of the algo
////////////////////////////////////////////////////////////////////////////////
StopWatchInterface  *uploadTimer, *downloadTimer, *bucketTimer, 
			 *mergeTimer, *totalTimer, *cpuTimer; 
////////////////////////////////////////////////////////////////////////////////
// Compare method for CPU sort
////////////////////////////////////////////////////////////////////////////////
inline int compare(const void *a, const void *b) {
	if(*((float *)a) < *((float *)b)) return -1; 
	else if(*((float *)a) > *((float *)b)) return 1; 
	else return 0; 
}
////////////////////////////////////////////////////////////////////////////////
// Forward declaration
////////////////////////////////////////////////////////////////////////////////
void cudaSort(float *origList, float minimum, float maximum,
			  float *resultList, int numElements);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

#ifdef GPUD
#ifdef THREADED
    #ifdef DRAM 
    #define NVMED_INIT(a)	nvmed_init(6)
    #define NVMED_SEND(a,b,c,d) nvmed_host_pipeline_send(a, b, c, d, NULL)
    #define NVMED_RECV(a,b,c,d) nvmed_host_pipeline_recv(a,b,c,d)
    #else
    #define NVMED_INIT(a)	nvmed_init(6)
    #define NVMED_SEND(a,b,c,d) nvmed_send_threaded(a,b,c,d)
    #define NVMED_RECV(a,b,c,d) nvmed_recv_threaded(a,b,c,d)
    #endif
#else
#define NVMED_INIT(a)	nvmed_init(a)
#define NVMED_SEND(a,b,c,d) nvmed_send(a,b,c,d)
#define NVMED_RECV(a,b,c,d) nvmed_recv(a,b,c,d)
#endif
#endif

int
main( int argc, char** argv)
{ 

  // Create timers for each sort
    struct timeval time_start, time_end, program_start, program_end;
    gettimeofday(&program_start, NULL);	

    sdkCreateTimer(&uploadTimer);
    sdkCreateTimer(&downloadTimer);
    sdkCreateTimer(&bucketTimer);
    sdkCreateTimer(&mergeTimer);
    sdkCreateTimer(&totalTimer);
    sdkCreateTimer(&cpuTimer);
	int numElements = 0;
    // Number of elements in the test bed
    	gettimeofday(&time_start, NULL);
#ifdef READING_FROM_BINARY
        numElements = atoi(argv[2]);
#else
       	if(strcmp(argv[1],"r") ==0) {
	numElements = SIZE; 
	}
	else {
		FILE *fp;
	fp = fopen(argv[1],"r");
	if(fp == NULL) {
	      cout << "Error reading file" << endl;
	      exit(EXIT_FAILURE);
	      }
	int count = 0;
	float c;

	while(fscanf(fp,"%f",&c) != EOF) {
	 count++;
}
	fclose(fp);
	numElements = count;
}
#endif

	cout << "Sorting list of " << numElements << " floats\n";
	// Generate random data
	// Memory space the list of random floats will take up
	int mem_size = numElements * sizeof(float); 
	// Allocate enough for the input list
	float *cpu_idata = (float *)malloc(mem_size);
	// Allocate enough for the output list on the cpu side
	float *cpu_odata = (float *)malloc(mem_size);
	// Allocate enough memory for the output list on the gpu side
	float *gpu_odata = (float *)malloc(mem_size);

	float datamin = FLT_MAX; 
	float datamax = -FLT_MAX; 
	if(strcmp(argv[1],"r")==0) {

	for (int i = 0; i < numElements; i++) {
	// Generate random floats between 0 and 1 for the input data
		cpu_idata[i] = ((float) rand() / RAND_MAX); 
	//Compare data at index to data minimum, if less than current minimum, set that element as new minimum
		datamin = min(cpu_idata[i], datamin);
	//Same as above but for maximum
		datamax = max(cpu_idata[i], datamax);
	}
}	else {
	FILE *fp;
#ifdef READING_FROM_BINARY
	fp = fopen(argv[1],"rb");
	fread(cpu_idata,sizeof(float),numElements,fp);
	fclose(fp);
   	gettimeofday(&time_end, NULL);
	printf("FileInput %lf seconds\n",((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec))/1000000.0);

    	gettimeofday(&time_start, NULL);	
	
/*	for(int i = 0; i < numElements; i++) {
	datamin = min(cpu_idata[i],datamin);
	datamax = max(cpu_idata[i],datamax);
	}
*/
	datamin = 0.0;
	datamax = 5.0;
//   	gettimeofday(&time_end, NULL);
//	printf("HGProfile: CPUFindMinMax %d m:%f M:%f\n",((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec)),datamin, datamax);
#else
	fp = fopen(argv[1],"r");
	for(int i = 0; i < numElements; i++) {
	fscanf(fp,"%f",&cpu_idata[i]);
	datamin = min(cpu_idata[i],datamin);
	datamax = max(cpu_idata[i],datamax);
	}
	fclose(fp);
   	gettimeofday(&time_end, NULL);
	printf("FileInput %lf seconds\n",((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec))/1000000.0);
    	gettimeofday(&time_start, NULL);	
#endif
	}
#ifndef CPU
	cout << "Sorting on GPU..." << flush; 
	// GPU Sort

	for (int i = 0; i < TEST; i++) 
		cudaSort(cpu_idata, datamin, datamax, gpu_odata, numElements);		
#endif

#ifdef VERIFY
	cout << "Sorting on CPU..." << flush; 
	// CPU Sort
	sdkStartTimer(&cpuTimer); 

	memcpy(cpu_odata, cpu_idata, mem_size);
        qsort(cpu_odata, numElements, sizeof(float), compare);
	sdkStopTimer(&cpuTimer); 
	cout << "done.\n";
/*	cout << "Checking result..." << flush; 
	// Result checking
	int count = 0; 
	for(int i = 0; i < numElements; i++)
		if(cpu_odata[i] != gpu_odata[i])
		{
			printf("Sort missmatch on element %d: \n", i); 
			printf("CPU = %f : GPU = %f\n", cpu_odata[i], gpu_odata[i]); 
			count++; 
			break; 
		}
	if(count == 0) cout << "PASSED.\n";
	else cout << "FAILED.\n";*/
#endif
	// Timer report
	printf("GPU iterations: %d\n", TEST); 
#ifdef TIMER
#ifdef VERIFY
	printf("Total CPU execution time: %lf seconds\n", sdkGetTimerValue(&cpuTimer)/1000);
#endif
#ifndef CPU
	printf("Total GPU execution time: %lf seconds\n", sdkGetTimerValue(&totalTimer) /1000);
	printf("    - Upload		: %lf seconds\n", sdkGetTimerValue(&uploadTimer) /1000);
	printf("    - Download		: %lf seconds\n", sdkGetTimerValue(&downloadTimer) /1000);
	printf("    - Bucket sort	: %lf seconds\n", sdkGetTimerValue(&bucketTimer) /1000);
	printf("    - Merge sort	: %lf seconds\n", sdkGetTimerValue(&mergeTimer) /1000);
#endif
#endif

#ifdef OUTPUT
    FILE *tp;
    const char filename2[]="./hybridoutput.txt";
    tp = fopen(filename2,"w");
    for(int i = 0; i < numElements; i++) {
        fprintf(tp,"%f ",cpu_idata[i]);
    }
    
    fclose(tp);
#endif
	
	// Release memory
   	gettimeofday(&program_end, NULL);
	printf("HGProfile: Total %lf\n",((program_end.tv_sec * 1000000 + program_end.tv_usec) - (program_start.tv_sec * 1000000 + program_start.tv_usec))-sdkGetTimerValue(&cpuTimer)/1000);
    sdkDeleteTimer(&uploadTimer);
    sdkDeleteTimer(&downloadTimer);
    sdkDeleteTimer(&bucketTimer);
    sdkDeleteTimer(&mergeTimer);
    sdkDeleteTimer(&totalTimer);
    sdkDeleteTimer(&cpuTimer);
#ifndef GPUD
	free(cpu_idata); 
#endif
	free(cpu_odata); free(gpu_odata); 
#ifdef GPUD
//	nvmed_deinit();
#endif
}


void cudaSort(float *origList, float minimum, float maximum,
			  float *resultList, int numElements)
{
	// Initialization and upload data
//    struct timeval time_start, time_end;
	float *d_input  = NULL; 
	float *d_output = NULL; 
	int mem_size = (numElements + DIVISIONS * 4) * sizeof(float); 
//   	gettimeofday(&time_start, NULL);
	sdkStartTimer(&uploadTimer);
	{

		hipMalloc((void**) &d_output, mem_size);
#ifdef GPUD
	#if TEST > 1
 	        hipMalloc((void**) &d_input, mem_size);
		hipMemcpy((void *) d_input, (void *)origList, numElements * sizeof(float),
				   hipMemcpyDeviceToDevice);
//                hipFree(origList);
	#else
                d_input=origList;
	#endif
#else
 	        hipMalloc((void**) &d_input, mem_size);
		hipMemcpy((void *) d_input, (void *)origList, numElements * sizeof(float),
				   hipMemcpyHostToDevice);
#endif
		init_bucketsort(numElements);
	}
//   	gettimeofday(&time_end, NULL);
//	printf("HGProfile: cudaMemcpyHD %d\n",((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec)));
	sdkStopTimer(&uploadTimer); 
    	//system("/homes/h1tseng/Gordon/PMCS/tools/powermeter/powermeter_client 0.0.0.0 \"e\" 27072");
    	//system("/homes/h1tseng/Gordon/PMCS/tools/powermeter/powermeter_client 0.0.0.0 \"b\" 27072");

	sdkStartTimer(&totalTimer); 

	// Bucketsort the list
	sdkStartTimer(&bucketTimer); 
		int *sizes = (int*) malloc(DIVISIONS * sizeof(int)); 
		int *nullElements = (int*) malloc(DIVISIONS * sizeof(int));  
		unsigned int *origOffsets = (unsigned int *) malloc((DIVISIONS + 1) * sizeof(int)); 
		bucketSort(d_input, d_output, numElements, sizes, nullElements, 
				   minimum, maximum, origOffsets); 
	sdkStopTimer(&bucketTimer); 

	// Mergesort the result
	sdkStartTimer(&mergeTimer); 
		float4 *d_origList = (float4*) d_output, 
		*d_resultList = (float4*) d_input;
		int newlistsize = 0; 
	
		for(int i = 0; i < DIVISIONS; i++)
			newlistsize += sizes[i] * 4;
		
		float4 *mergeresult = runMergeSort(	newlistsize, DIVISIONS, d_origList, d_resultList, 
			sizes, nullElements, origOffsets); //d_origList; 
		hipDeviceSynchronize(); 
	sdkStopTimer(&mergeTimer); 
	sdkStopTimer(&totalTimer); 
//	fprintf(stderr, "%p %p\n",resultList, mergeresult );
	// Download result
	sdkStartTimer(&downloadTimer); 
/*		checkCudaErrors(	hipMemcpy((void *) resultList, 
				(void *)mergeresult, numElements * sizeof(float), hipMemcpyDeviceToHost) );*/
	sdkStopTimer(&downloadTimer); 

	// Clean up
	finish_bucketsort(); 
//#ifdef GPUD
	hipFree(d_input); 
//#endif
	hipFree(d_output); 
	free(nullElements); free(sizes); 
}
